// TODO:
// - backprop
// - python bindings
// - ...

#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

hipError_t err;

__global__ void vectorAddInPlace(float* base, float* addMe, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n)
        base[tid] += addMe[tid];
}


__global__ void matrixMulShared(
    float *a,
    float *b,
    float *out,
    int M,
    int N,
    int K
) {
    __shared__ float sharedA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sharedB[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float sum = 0.0f;

    for (int t = 0; t < (K - 1) / TILE_WIDTH + 1; ++t) {
        if (row < M && t * TILE_WIDTH + tx < K)
            sharedA[ty][tx] = a[row * K + t * TILE_WIDTH + tx];
        else
            sharedA[ty][tx] = 0.0f;

        if (col < N && t * TILE_WIDTH + ty < K)
            sharedB[ty][tx] = b[(t * TILE_WIDTH + ty) * N + col];
        else
            sharedB[ty][tx] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
            sum += sharedA[ty][k] * sharedB[k][tx];

        __syncthreads();
    }

    if (row < M && col < N)
        out[row * N + col] = sum;
}


void initMatrixRandom(float *mat, int n_elem) {
    for (int i = 0; i < n_elem; ++i)
        mat[i] = rand() / (float)RAND_MAX;
}


void initMatrixZeros(float *mat, int n_elem) {
    for (int i = 0; i < n_elem; ++i)
        mat[i] = 0;
}


void linear(
    int inDim,
    int outDim,
    float* d_in,
    float* d_W,
    float* d_b,
    float* d_out
) {
    dim3 dimGrid((outDim + TILE_WIDTH - 1) / TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    matrixMulShared<<<dimGrid, dimBlock>>>(d_in, d_W, d_out, 1, outDim, inDim);

    int threadsPerBlock = 256; // A common choice, can be adjusted based on your GPU
    int blocks = (outDim + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddInPlace<<<blocks, threadsPerBlock>>>(d_out, d_b, outDim);
}

typedef struct NN {
    int sizeIn;
    float* d_W1;
    float* d_b1;
    int size1;
    float* d_W2;
    float* d_b2;
    int size2;
    float* d_W3;
    float* d_b3;
    int sizeOut;
} NN;


void initLayer(int sizeIn, int sizeOut, float** d_W, float** d_b) {
    size_t sizeW = sizeof(float) * sizeIn * sizeOut;
    size_t sizeB = sizeof(float) * sizeOut;

    float* h_W = (float*)malloc(sizeW);
    initMatrixRandom(h_W, sizeIn * sizeOut);

    err = hipMalloc(d_W, sizeW);
    if (err != hipSuccess) {
        printf("error `cudaMalloc`ing d_W. error: %s\n", hipGetErrorString(err));
    }

    err = hipMemcpy(*d_W, h_W, sizeW, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("error `cudaMemcpy`ing to d_W. error: %s\n", hipGetErrorString(err));
    }

    float* h_b = (float*)malloc(sizeB);
    initMatrixZeros(h_b, sizeOut);

    err = hipMalloc(d_b, sizeB);
    if (err != hipSuccess) {
        printf("error `cudaMalloc`ing d_b. error: %s\n", hipGetErrorString(err));
    }

    err = hipMemcpy(*d_b, h_b, sizeB, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("error `cudaMemcpy`ing to d_b. error: %s\n", hipGetErrorString(err));
    }
}


void printDeviceArr(float* d_x, int n_elem, char* name) {
    size_t size = sizeof(float) * n_elem;
    float* h_x = (float*)malloc(size);
    err = hipMemcpy(h_x, d_x, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("error `cudaMemcpy`ing to d_b. error: %s\n", hipGetErrorString(err));
    }
    printf("%s:\n", name);
    for (int i = 0; i < n_elem; ++i)
        printf("%f ", h_x[i]);
    printf("\n\n");  
    free(h_x);
}


NN* createNN(
    int sizeIn,
    int sizeLayer1,
    int sizeLayer2,
    int sizeOut
) {
    NN* nn = (NN*)malloc(sizeof(NN));

    nn->sizeIn = sizeIn;
    nn->size1 = sizeLayer1;
    nn->size2 = sizeLayer2;

    initLayer(sizeIn, sizeLayer1, &nn->d_W1, &nn->d_b1);
    initLayer(sizeLayer1, sizeLayer2, &nn->d_W2, &nn->d_b2);
    initLayer(sizeLayer2, sizeOut, &nn->d_W3, &nn->d_b3);
    printDeviceArr(nn->d_W1, nn->sizeIn * nn->size1, (char*)"d_W1");
    printDeviceArr(nn->d_W2, nn->size1 * nn->size2, (char*)"d_W2");

    nn->sizeOut = sizeOut;
    return nn;
}


void freeNN(NN* nn) {
    free(nn->d_W1);
    free(nn->d_b1);
    free(nn->d_W2);
    free(nn->d_b2);
    free(nn->d_W3);
    free(nn->d_b3);
    free(nn);
}


void runNN(
    NN* nn,
    float* d_in,
    float* d_out
) {
    float *d_h1, *d_h2;

    err = hipMalloc(&d_h1, sizeof(float) * nn->size1);
    if (err != hipSuccess) {
        printf("error `cudaMemcpy`ing to d_b. error: %s\n", hipGetErrorString(err));
    }

    err = hipMalloc(&d_h2, sizeof(float) * nn->size2);
    if (err != hipSuccess) {
        printf("error `cudaMemcpy`ing to d_b. error: %s\n", hipGetErrorString(err));
    }

    linear(nn->sizeIn, nn->size1, d_in, nn->d_W1, nn->d_b1, d_h1);
    linear(nn->size1, nn->size2, d_h1, nn->d_W2, nn->d_b2, d_h2);
    linear(nn->size2, nn->sizeOut, d_h2, nn->d_W3, nn->d_b3, d_out);

    hipFree(d_h1);
    hipFree(d_h2);
}


int main() {
    int inDim = 3;
    int hiddenDim1 = 5;
    int hiddenDim2 = 3;
    int outDim = 10;

    NN* nn = createNN(inDim, hiddenDim1, hiddenDim2, outDim);
    
    size_t sizeIn = sizeof(float) * inDim;

    float* h_in = (float*)malloc(sizeIn);
    initMatrixRandom(h_in, inDim);

    float* d_in;
    err = hipMalloc(&d_in, sizeIn);
    if (err != hipSuccess) {
        printf("error `cudaMalloc`ing d_in. error: %s\n", hipGetErrorString(err));
    }

    err = hipMemcpy(d_in, h_in, sizeIn, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("error `cudaMemcpy`ing h_in to d_in. error: %s\n", hipGetErrorString(err));
    }

    size_t sizeOut = sizeof(float) * outDim;

    float* d_out;
    err = hipMalloc(&d_out, sizeOut);
    if (err != hipSuccess) {
        printf("error `cudaMalloc`ing d_out. error: %s\n", hipGetErrorString(err));
    }

    runNN(nn, d_in, d_out);

    float* h_out = (float*)malloc(sizeOut);
    err = hipMemcpy(h_out, d_out, sizeOut, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("error `cudaMemcpy`ing d_out to h_out. error: %s\n", hipGetErrorString(err));
    }

    // ==========

    printf("First few elements of `out`:\n");
    for (int i = 0; i < outDim; ++i)
        printf("%f ", h_out[i]);

}
